#include "hip/hip_runtime.h"


#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <math.h>

#define MAX_NAME 256 /* tamanho maximo para nome de arquivo */
#define PI 3.14159265359

double **matrizpesos;

struct param{
	int totalthreads;
	int linhas;
	int colunas;
	int tamanho; //se a divisao nao for inteira aqui vai o mais 1
	int sobra; //resto da divisao que vamos distribuir
	int raio; //distancia para considerar os pixels
};

void InicializaMatrizPesos(int raio);
double **AlocaMatrizDouble(int lin, int col);
double **LiberaMatrizDouble(int lin, int col, double **mat);
int **AlocaMatriz(int lin, int col);
int **LiberaMatriz(int lin, int col, int **mat);

__global__ void filtra(int **cudaOldimage, int **cudaNewimage, double **cudaMatrizPesos, param parameters){
	int linhai, linhaf, deslocamento, meutam, index;

	index = threadIdx.x + blockIdx.x * blockDim.x; //index global da thread
	meutam = parameters.tamanho;
	deslocamento = 0;

	if (parameters.sobra > 0){
		if (index < parameters.sobra){
			meutam++;
		}
		else {
			deslocamento = parameters.sobra;
		}
	}
	linhai = (meutam)* index + deslocamento;
	if (index == parameters.totalthreads - 1){
		linhaf = parameters.linhas - 1;
	}
	else {
		linhaf = (meutam)+linhai - 1;
	}
	//printf("Eu sou a thread %d fico com: linha %d a %d\n", index, linhai, linhaf);

	//tratamento_normal(linhai, linhaf);
	int dls, dli, dce, dcd;
	int r, g, b, nr, ng, nb;
	int l;
	int c;
	int lin_mat_pes, col_mat_pes;
	double acumular, acumulag, acumulab;
	for (l = linhai; l <= linhaf; l++){
		if ((l - parameters.raio) < 0){
			dls = parameters.raio - l;
		}
		else dls = 0;
		if ((l + parameters.raio) >= parameters.linhas){
			dli = l + parameters.raio - (parameters.linhas - 1);
		}
		else dli = 0;
		//printf("Linha: %d ... dls(%d) e dli(%d)\n", l, dls, dli);
		for (c = 0; c < parameters.colunas; c++){
			acumular = 0;
			acumulag = 0;
			acumulab = 0;
			if ((c - parameters.raio) < 0){
				dce = parameters.raio - c;
			}
			else dce = 0;
			if ((c + parameters.raio) >= parameters.colunas){
				dcd = c + parameters.raio - (parameters.colunas - 1);
			}
			else dcd = 0;
			for (lin_mat_pes = dls; lin_mat_pes < (parameters.raio * 2 + 1 - dli); lin_mat_pes++){
				for (col_mat_pes = dce; col_mat_pes < (parameters.raio * 2 + 1 - dcd); col_mat_pes++){
					r = cudaOldimage[l - parameters.raio + lin_mat_pes][c - parameters.raio + col_mat_pes] / 1000000;
					g = (cudaOldimage[l - parameters.raio + lin_mat_pes][c - parameters.raio + col_mat_pes] - r * 1000000) / 1000;;
					b = cudaOldimage[l - parameters.raio + lin_mat_pes][c - parameters.raio + col_mat_pes] - r * 1000000 - g * 1000;
					acumular += (r * cudaMatrizPesos[lin_mat_pes][col_mat_pes]);
					acumulag += (g * cudaMatrizPesos[lin_mat_pes][col_mat_pes]);
					acumulab += (b * cudaMatrizPesos[lin_mat_pes][col_mat_pes]);
				}
			}
			nr = acumular;
			ng = acumulag;
			nb = acumulab;
			cudaNewimage[l][c] = nr * 1000000 + ng * 1000 + nb;
		}
	}

}

int main() {
	FILE *arqin;
	FILE *arqout;
	char narqin[MAX_NAME] = "c:\\temp\\reddead.ppm";
	char narqout[MAX_NAME] = "c:\\temp\\reddead2.ppm";
	char key[128];
	int i, j, max, r, g, b; //auxiliares
	int Blocks = 1;
	int ThreadsPerBlock = 1;

	struct param parameters;
	hipError_t cudaStatus;

	parameters.totalthreads = Blocks * ThreadsPerBlock;

	printf("Qual raio?\n");
	scanf("%d", &parameters.raio);

	printf("Arquivo de entrada: %s\n", narqin);
	arqin = fopen(narqin, "r");

	if (arqin == NULL) {
		printf("Erro na abertura do arquivo %s\n", narqin);
		return 1;
	}

	printf("Arquivo de saida: %s\n", narqout);
	arqout = fopen(narqout, "w");

	if (arqout == NULL) {
		printf("Erro na abertura do arquivo %s\n", narqin);
		return 1;
	}

	fscanf(arqin, "%s", key);//leio cabe�alho
	fprintf(arqout, "%s\n", key);//j� escrevo o cabe�alho no novo arquivo
	printf("Arquivo tipo: %s \n", key);
	fscanf(arqin, "%d %d %d", &parameters.colunas, &parameters.linhas, &max);//leio mais dados do cabe�alho
	fprintf(arqout, "%d %d \n%d", parameters.colunas, parameters.linhas, max);//j� escrevo esses dados no novo arquivo
	printf("Colunas = %d \nLinhas = %d \n", parameters.colunas, parameters.linhas);

	//vamos definir o tamanho para cada um
	parameters.tamanho = parameters.linhas / parameters.totalthreads;
	if ((parameters.linhas % parameters.totalthreads) > 0){
		parameters.sobra = parameters.linhas % parameters.totalthreads;
	}
	else {
		parameters.sobra = 0;
	}

	printf("Tamanho %d\n", parameters.tamanho);

	//por enquanto nao vamos aceitar imagem com apenas uma linha
	if (parameters.linhas < parameters.totalthreads){
		printf("Mais threads do que dados %s\n", narqin);
		return 0;
	}
	
	int **oldimage = AlocaMatriz(parameters.linhas, parameters.colunas);
	int **newimage = AlocaMatriz(parameters.linhas, parameters.colunas);
	matrizpesos = AlocaMatrizDouble(parameters.raio * 2 + 1, parameters.raio * 2 + 1);
	InicializaMatrizPesos(parameters.raio);
	
	for (i = 0; i <= parameters.linhas - 1; i++)
		for (j = 0; j <= parameters.colunas - 1; j++) {
		fscanf(arqin, " %d %d %d ", &r, &g, &b);
		//printf("RGB: %d %d %d \n", r, g, b);
		oldimage[i][j] = r * 1000000 + g * 1000 + b;
		/*
		rgb = oldimage[i][j];
		nr = rgb/1000000;
		ng = (rgb-r*1000000)/1000;
		nb = rgb-r*1000000-g*1000;
		if ((nr != r) || (ng != g) || (nb != b)) printf("errooou");
		printf("Valor: %d\n", rgb);
		printf("Valor R: %d\n", r);
		printf("Valor G: %d\n", g);
		printf("Valor B: %d\n", b);
		*/
		}

	// aloca a mem�ria no device
	int size_m_int = parameters.linhas*parameters.colunas*sizeof(int);	// tamanho da mem�ria que ser� aolocado para as matrizes
	int size_m_double = (parameters.raio * 2 + 1) * (parameters.raio * 2 + 1) * sizeof(double);	// tamanho da mem�ria que ser� aolocado para a matriz de pesos
	int **doldimage, **dnewimage;
	double **dmatrizpesos;

	printf("1...alocando doldimage na GPU...");
	cudaStatus = hipMalloc((void**)&doldimage, size_m_int);
	if (cudaStatus != hipSuccess) {
		printf("hipMalloc failed!");
		return 1;
	}
	printf("2...alocando dnewimage na GPU...");
	cudaStatus = hipMalloc((void**)&dnewimage, size_m_int);
	if (cudaStatus != hipSuccess) {
		printf("hipMalloc failed!");
		return 1;
	}
	printf("3...alocando dmatrizpesos na GPU...");
	cudaStatus = hipMalloc((void**)&dmatrizpesos, size_m_double);
	if (cudaStatus != hipSuccess) {
		printf("hipMalloc failed!");
		return 1;
	}

	// copia as matrizes do host para o device
	printf("4...copiando oldimage para GPU...");
	cudaStatus = hipMemcpy(doldimage, oldimage, size_m_int, hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		printf("hipMemcpy failed!");
		return 1;
	}
	printf("5...copiando matrizpesos para GPU...");
	cudaStatus = hipMemcpy(dmatrizpesos, matrizpesos, size_m_double, hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		printf("hipMemcpy failed!");
		return 1;
	}

	//executa o kernel
	printf("6...rodando kernel...");
	filtra<<<Blocks, ThreadsPerBlock>>>(doldimage, dnewimage, dmatrizpesos, parameters);

	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		printf("addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		return 1;
	}

	// cudaDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	printf("7...sincronizando com device...");
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		printf("hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		return 1;
	}

	//copia matriz resultante da GPU para a CPU
	printf("8...copiando dnewimage da GPU para processador...");
	cudaStatus = hipMemcpy(newimage, dnewimage, size_m_int, hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		printf("hipMemcpy failed!");
		return 1;
	}

	//escrever novo arquivo    
	printf("9...escrevendo nova imagem...");
	for (i = 0; i <= parameters.linhas - 1; i++){
		fprintf(arqout, "\n");
		for (j = 0; j <= parameters.colunas - 1; j++) {
			r = newimage[i][j] / 1000000;
			g = (newimage[i][j] - r * 1000000) / 1000;
			b = newimage[i][j] - r * 1000000 - g * 1000;
			fprintf(arqout, "%d %d %d ", r, g, b);
		}
	}

	//    for (i = 0; i <= linhas - 1; i++) for (j = 0; j <= colunas - 1; j++) printf("RGB: %d %d %d \n", newimage[i][j*3], newimage[i][j*3+1], newimage[i][j*3+2]);

	printf("10...liberando matrizes...");
	LiberaMatriz(parameters.linhas, parameters.colunas, oldimage);
	LiberaMatriz(parameters.linhas, parameters.colunas, newimage);
	LiberaMatrizDouble(parameters.raio * 2 + 1, parameters.raio * 2 + 1, matrizpesos);
	hipFree(doldimage);
	hipFree(dnewimage);
	hipFree(dmatrizpesos);

	fclose(arqin);
	fclose(arqout);

	printf("Fim programa.\n");
	return 0;
}

void InicializaMatrizPesos(int raio){
	int i, j;
	double e, g;
	double somapesos = 0;
	float sigma = raio;
	for (i = 0; i < sigma * 2 + 1; i++){
		//printf("\n");
		for (j = 0; j < raio * 2 + 1; j++){
			e = pow((float)exp(1.0), ((-1)*(pow((i - sigma), 2) + pow((j - sigma), 2)) / (2 * pow(sigma, 2))));
			//printf("P(%d,%d)\n", i, j);
			//printf("E = %.4f - PARTEDECIMA = %.4f\n", e, partedecima);
			g = e / (2 * PI*pow(sigma, 2));
			matrizpesos[i][j] = g;
			somapesos += g;
			//printf("P(%d,%d) = %.4f ;", i, j, g);
		}
	}
	for (i = 0; i < sigma * 2 + 1; i++){
		//printf("de novo \n");
		for (j = 0; j < raio * 2 + 1; j++){
			matrizpesos[i][j] = matrizpesos[i][j] / somapesos;
			//printf("P(%d,%d) = %.4f ;", i, j, matrizpesos[i][j]);
		}
	}
	//printf("somapesos = %.5f\n", somapesos);
}

int **AlocaMatriz(int lin, int col){
	int **mat;  /* ponteiro para a matriz */
	int i;    /* variavel auxiliar      */
	if (lin < 1 || col < 1) { /* verifica parametros recebidos */
		printf("** Erro: Parametro invalido **\n");
		return(NULL);
	}
	/* aloca as linhas da matriz */
	mat = (int **)calloc(lin, sizeof(int *));
	if (mat == NULL) {
		printf("** Erro: Memoria Insuficiente **");
		return(NULL);
	}
	/* aloca as colunas da matriz */
	for (i = 0; i < lin; i++){
		mat[i] = (int*)calloc(col, sizeof(int));
		if (mat[i] == NULL) {
			printf("** Erro: Memoria Insuficiente **");
			return(NULL);
		}
	}
	return(mat); /* retorna o ponteiro para a matriz */
}

int **LiberaMatriz(int lin, int col, int **mat){
	int i;  /* variavel auxiliar */
	if (mat == NULL) return(NULL);
	if (lin < 1 || col < 1){  /* verifica parametros recebidos */
		printf("** Erro: Parametro invalido **\n");
		return(mat);
	}
	for (i = 0; i<lin; i++) free(mat[i]); /* libera as linhas da matriz */
	free(mat);      /* libera a matriz */
	return(NULL); /* retorna um ponteiro nulo */
}

double **AlocaMatrizDouble(int lin, int col){
	double **mat;  /* ponteiro para a matriz */
	int i;    /* variavel auxiliar      */
	if (lin < 1 || col < 1) { /* verifica parametros recebidos */
		printf("** Erro: Parametro invalido **\n");
		return(NULL);
	}
	/* aloca as linhas da matriz */
	mat = (double **)calloc(lin, sizeof(double *));
	if (mat == NULL) {
		printf("** Erro: Memoria Insuficiente **");
		return(NULL);
	}
	/* aloca as colunas da matriz */
	for (i = 0; i < lin; i++){
		mat[i] = (double*)calloc(col, sizeof(double));
		if (mat[i] == NULL) {
			printf("** Erro: Memoria Insuficiente **");
			return(NULL);
		}
	}
	return(mat); /* retorna o ponteiro para a matriz */
}

double **LiberaMatrizDouble(int lin, int col, double **mat){
	int i;  /* variavel auxiliar */
	if (mat == NULL) return(NULL);
	if (lin < 1 || col < 1){  /* verifica parametros recebidos */
		printf("** Erro: Parametro invalido **\n");
		return(mat);
	}
	for (i = 0; i<lin; i++) free(mat[i]); /* libera as linhas da matriz */
	free(mat);      /* libera a matriz */
	return(NULL); /* retorna um ponteiro nulo */
}